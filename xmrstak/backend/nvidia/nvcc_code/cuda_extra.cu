#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <sstream>
#include <algorithm>
#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include  <algorithm>
#include "xmrstak/jconf.hpp"


typedef unsigned char BitSequence;
typedef unsigned long long DataLength;

#include "xmrstak/backend/cryptonight.hpp"
#include "cryptonight.hpp"
#include "cuda_extra.hpp"
#include "cuda_keccak.hpp"
#include "cuda_blake.hpp"
#include "cuda_groestl.hpp"
#include "cuda_jh.hpp"
#include "cuda_skein.hpp"
#include "cuda_device.hpp"
#include "cuda_aes.hpp"

__constant__ uint8_t d_sub_byte[16][16] ={
	{0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76 },
	{0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0 },
	{0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15 },
	{0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75 },
	{0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84 },
	{0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf },
	{0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8 },
	{0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2 },
	{0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73 },
	{0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb },
	{0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79 },
	{0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08 },
	{0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a },
	{0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e },
	{0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf },
	{0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16 }
};

__device__ __forceinline__ void cryptonight_aes_set_key( uint32_t * __restrict__ key, const uint32_t * __restrict__ data )
{
	int i, j;
	uint8_t temp[4];
	const uint32_t aes_gf[] = { 0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36 };

	MEMSET4( key, 0, 40 );
	MEMCPY4( key, data, 8 );

#pragma unroll
	for ( i = 8; i < 40; i++ )
	{
		*(uint32_t *) temp = key[i - 1];
		if ( i % 8 == 0 )
		{
			*(uint32_t *) temp = ROTR32( *(uint32_t *) temp, 8 );
			for ( j = 0; j < 4; j++ )
				temp[j] = d_sub_byte[( temp[j] >> 4 ) & 0x0f][temp[j] & 0x0f];
			*(uint32_t *) temp ^= aes_gf[i / 8 - 1];
		}
		else
		{
			if ( i % 8 == 4 )
			{
#pragma unroll
				for ( j = 0; j < 4; j++ )
					temp[j] = d_sub_byte[( temp[j] >> 4 ) & 0x0f][temp[j] & 0x0f];
			}
		}

		key[i] = key[( i - 8 )] ^ *(uint32_t *) temp;
	}
}

__device__ __forceinline__ void mix_and_propagate( uint32_t* state )
{
	uint32_t tmp0[4];
	for(size_t x = 0; x < 4; ++x)
		tmp0[x] = (state)[x];

	// set destination [0,6]
	for(size_t t = 0; t < 7; ++t)
		for(size_t x = 0; x < 4; ++x)
			(state + 4 * t)[x] = (state + 4 * t)[x] ^ (state + 4 * (t + 1))[x];

	// set destination 7
	for(size_t x = 0; x < 4; ++x)
		(state + 4 * 7)[x] = (state + 4 * 7)[x] ^ tmp0[x];
}

template<xmrstak_algo_id ALGO>
__global__ void cryptonight_extra_gpu_prepare( int threads, uint32_t * __restrict__ d_input, uint32_t len, uint32_t startNonce, uint32_t * __restrict__ d_ctx_state, uint32_t * __restrict__ d_ctx_state2, uint32_t * __restrict__ d_ctx_a, uint32_t * __restrict__ d_ctx_b, uint32_t * __restrict__ d_ctx_key1, uint32_t * __restrict__ d_ctx_key2 )
{
	int thread = ( blockDim.x * blockIdx.x + threadIdx.x );
	__shared__ uint32_t sharedMemory[1024];

	if(ALGO == cryptonight_heavy || ALGO == cryptonight_haven || ALGO == cryptonight_bittube2 || ALGO == cryptonight_superfast)
	{
		cn_aes_gpu_init( sharedMemory );
		__syncthreads( );
	}
	if ( thread >= threads )
		return;

	uint32_t ctx_state[50];
	uint32_t ctx_a[4];
	uint32_t ctx_b[4];
	uint32_t ctx_key1[40];
	uint32_t ctx_key2[40];
	uint32_t input[21];

	memcpy( input, d_input, len );
	//*((uint32_t *)(((char *)input) + 39)) = startNonce + thread;
	uint32_t nonce = startNonce + thread;
	for ( int i = 0; i < sizeof (uint32_t ); ++i )
		( ( (char *) input ) + 39 )[i] = ( (char*) ( &nonce ) )[i]; //take care of pointer alignment

	cn_keccak( (uint8_t *) input, len, (uint8_t *) ctx_state );
	cryptonight_aes_set_key( ctx_key1, ctx_state );
	cryptonight_aes_set_key( ctx_key2, ctx_state + 8 );

	XOR_BLOCKS_DST( ctx_state, ctx_state + 8, ctx_a );
	XOR_BLOCKS_DST( ctx_state + 4, ctx_state + 12, ctx_b );
	memcpy( d_ctx_a + thread * 4, ctx_a, 4 * 4 );
	if(ALGO == cryptonight_monero_v8 || ALGO == cryptonight_v8_reversewaltz)
	{
		memcpy( d_ctx_b + thread * 16, ctx_b, 4 * 4 );
		// bx1
		XOR_BLOCKS_DST( ctx_state + 16, ctx_state + 20, ctx_b );
		memcpy( d_ctx_b + thread * 16 + 4, ctx_b, 4 * 4 );
		// division_result
		memcpy( d_ctx_b + thread * 16 + 2 * 4, ctx_state + 24, 4 * 2 );
		// sqrt_result
		memcpy( d_ctx_b + thread * 16 + 2 * 4 + 2, ctx_state + 26, 4 * 2 );
	}
	else if(ALGO == cryptonight_r_wow || ALGO == cryptonight_r)
	{
		memcpy(d_ctx_b + thread * 16, ctx_b, 4 * 4);
		// bx1
		XOR_BLOCKS_DST(ctx_state + 16, ctx_state + 20, ctx_b);
		memcpy(d_ctx_b + thread * 16 + 4, ctx_b, 4 * 4);
		// r0, r1, r2, r3
		memcpy(d_ctx_b + thread * 16 + 2 * 4, ctx_state + 24, 4 * 8);
	}
	else
		memcpy( d_ctx_b + thread * 4, ctx_b, 4 * 4 );

	memcpy( d_ctx_key1 + thread * 40, ctx_key1, 40 * 4 );
	memcpy( d_ctx_key2 + thread * 40, ctx_key2, 40 * 4 );
	memcpy( d_ctx_state + thread * 50, ctx_state, 50 * 4 );

	if(ALGO == cryptonight_heavy || ALGO == cryptonight_haven || ALGO == cryptonight_bittube2 || ALGO == cryptonight_superfast)
	{

		for(int i=0; i < 16; i++)
		{
			for(size_t t = 4; t < 12; ++t)
			{
				cn_aes_pseudo_round_mut( sharedMemory, ctx_state + 4u * t, ctx_key1 );
			}
			// scipt first 4 * 128bit blocks = 4 * 4 uint32_t values
			mix_and_propagate(ctx_state + 4 * 4);
		}
		// double buffer to move manipulated state into phase1
		memcpy( d_ctx_state2 + thread * 50, ctx_state, 50 * 4 );
	}
}

template<xmrstak_algo_id ALGO>
__global__ void cryptonight_extra_gpu_final( int threads, uint64_t target, uint32_t* __restrict__ d_res_count, uint32_t * __restrict__ d_res_nonce, uint32_t * __restrict__ d_ctx_state,uint32_t * __restrict__ d_ctx_key2 )
{
	const int thread = blockDim.x * blockIdx.x + threadIdx.x;

	__shared__ uint32_t sharedMemory[1024];

	if(ALGO == cryptonight_gpu || ALGO == cryptonight_heavy || ALGO == cryptonight_haven ||
		ALGO == cryptonight_bittube2 || ALGO == cryptonight_superfast)
	{
		cn_aes_gpu_init( sharedMemory );
		__syncthreads( );
	}
	if ( thread >= threads )
		return;

	int i;
	uint32_t * __restrict__ ctx_state = d_ctx_state + thread * 50;
	uint64_t hash[4];
	uint32_t state[50];

	#pragma unroll
	for ( i = 0; i < 50; i++ )
		state[i] = ctx_state[i];

	if(ALGO == cryptonight_gpu || ALGO == cryptonight_heavy || ALGO == cryptonight_haven ||
		ALGO == cryptonight_bittube2 || ALGO == cryptonight_superfast)
	{
		uint32_t key[40];

		// load keys
		MEMCPY8( key, d_ctx_key2 + thread * 40, 20 );

		for(int i=0; i < 16; i++)
		{
			for(size_t t = 4; t < 12; ++t)
			{
				cn_aes_pseudo_round_mut( sharedMemory, state + 4u * t, key );
			}
			// scipt first 4 * 128bit blocks = 4 * 4 uint32_t values
			mix_and_propagate(state + 4 * 4);
		}
	}
	cn_keccakf2( (uint64_t *) state );

	if(ALGO == cryptonight_gpu)
	{
		if ( ((uint64_t*)state)[3] < target )
		{
			uint32_t idx = atomicInc( d_res_count, 0xFFFFFFFF );

			if(idx < 10)
				d_res_nonce[idx] = thread;
		}
	}
	else
	{
		switch ( ( (uint8_t *) state )[0] & 0x03 )
		{
		case 0:
			cn_blake( (const uint8_t *) state, 200, (uint8_t *) hash );
			break;
		case 1:
			cn_groestl( (const BitSequence *) state, 200, (BitSequence *) hash );
			break;
		case 2:
			cn_jh( (const BitSequence *) state, 200, (BitSequence *) hash );
			break;
		case 3:
			cn_skein( (const BitSequence *) state, 200, (BitSequence *) hash );
			break;
		default:
			break;
		}

		// Note that comparison is equivalent to subtraction - we can't just compare 8 32-bit values
		// and expect an accurate result for target > 32-bit without implementing carries

		if ( hash[3] < target )
		{
			uint32_t idx = atomicInc( d_res_count, 0xFFFFFFFF );

			if(idx < 10)
				d_res_nonce[idx] = thread;
		}
	}
}

extern "C" void cryptonight_extra_cpu_set_data( nvid_ctx* ctx, const void *data, uint32_t len )
{
	ctx->inputlen = len;
	CUDA_CHECK(ctx->device_id, hipMemcpy( ctx->d_input, data, len, hipMemcpyHostToDevice ));
}

extern "C" int cryptonight_extra_cpu_init(nvid_ctx* ctx)
{
	CU_CHECK(ctx->device_id, hipDeviceGet(&ctx->cuDevice, ctx->device_id));
    CU_CHECK(ctx->device_id, hipCtxCreate(&ctx->cuContext, 0, ctx->cuDevice));

	hipError_t err;
	err = hipSetDevice(ctx->device_id);
	if(err != hipSuccess)
	{
		printf("GPU %d: %s", ctx->device_id, hipGetErrorString(err));
		return 0;
	}

	CUDA_CHECK(ctx->device_id, hipDeviceReset());
	switch(ctx->syncMode)
	{
	case 0:
		CUDA_CHECK(ctx->device_id, hipSetDeviceFlags(hipDeviceScheduleAuto));
		break;
	case 1:
		CUDA_CHECK(ctx->device_id, hipSetDeviceFlags(hipDeviceScheduleSpin));
		break;
	case 2:
		CUDA_CHECK(ctx->device_id, hipSetDeviceFlags(hipDeviceScheduleYield));
		break;
	case 3:
		CUDA_CHECK(ctx->device_id, hipSetDeviceFlags(hipDeviceScheduleBlockingSync));
		break;

	};

	// prefer shared memory over L1 cache
	CUDA_CHECK(ctx->device_id, hipDeviceSetCacheConfig(hipFuncCachePreferShared));

	auto neededAlgorithms = ::jconf::inst()->GetCurrentCoinSelection().GetAllAlgorithms();

	size_t hashMemSize = 0;
	for(const auto algo : neededAlgorithms)
	{
		hashMemSize = std::max(hashMemSize, algo.Mem());
	}

	size_t wsize = ctx->device_blocks * ctx->device_threads;
	CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_ctx_state, 50 * sizeof(uint32_t) * wsize));
	size_t ctx_b_size = 4 * sizeof(uint32_t) * wsize;
	if(
		std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_heavy) != neededAlgorithms.end() ||
		std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_haven) != neededAlgorithms.end() ||
		std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_bittube2) != neededAlgorithms.end() ||
		std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_superfast) != neededAlgorithms.end()
	)
	{
		// extent ctx_b to hold the state of idx0
		ctx_b_size += sizeof(uint32_t) * wsize;
		// create a double buffer for the state to exchange the mixed state to phase1
		CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_ctx_state2, 50 * sizeof(uint32_t) * wsize));
	}
	else if(std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_conceal) != neededAlgorithms.end())
	{
		ctx_b_size += sizeof(uint32_t) * 4 * wsize;
	}
	else if((std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_monero_v8) != neededAlgorithms.end())
		|| (std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_v8_reversewaltz) != neededAlgorithms.end()))
	{
		// bx0 (16byte), bx1 (16byte), division_result (8byte) and sqrt_result (8byte), padding (16byte)
		ctx_b_size = 4 * 4 * sizeof(uint32_t) * wsize;
	}
	else if(
		std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_r) != neededAlgorithms.end() ||
		std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_r_wow) != neededAlgorithms.end()
	)
	{
		// bx0 (16byte), bx1 (16byte), and [r0, r1, r2, r3] (a 8byte)
		ctx_b_size = 4 * 4 * sizeof(uint32_t) * wsize;
	}
	else
		ctx->d_ctx_state2 = ctx->d_ctx_state;

	CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_ctx_key1, 40 * sizeof(uint32_t) * wsize));
	CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_ctx_key2, 40 * sizeof(uint32_t) * wsize));
	CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_ctx_text, 32 * sizeof(uint32_t) * wsize));
	CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_ctx_a, 4 * sizeof(uint32_t) * wsize));
	CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_ctx_b, ctx_b_size));
	// POW block format http://monero.wikia.com/wiki/PoW_Block_Header_Format
	CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_input, 21 * sizeof (uint32_t ) ));
	CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_result_count, sizeof (uint32_t ) ));
	CUDA_CHECK(ctx->device_id, hipMalloc(&ctx->d_result_nonce, 10 * sizeof (uint32_t ) ));
	CUDA_CHECK_MSG(
		ctx->device_id,
		"\n**suggestion: Try to reduce the value of the attribute 'threads' in the NVIDIA config file.**",
		hipMalloc(&ctx->d_long_state, hashMemSize * wsize));
	return 1;
}

extern "C" void cryptonight_extra_cpu_prepare(nvid_ctx* ctx, uint32_t startNonce, const xmrstak_algo& miner_algo)
{
	int threadsperblock = 128;
	uint32_t wsize = ctx->device_blocks * ctx->device_threads;

	dim3 grid( ( wsize + threadsperblock - 1 ) / threadsperblock );
	dim3 block( threadsperblock );

	if(miner_algo == cryptonight_heavy)
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_extra_gpu_prepare<cryptonight_heavy><<<grid, block >>>( wsize, ctx->d_input, ctx->inputlen, startNonce,
			ctx->d_ctx_state,ctx->d_ctx_state2, ctx->d_ctx_a, ctx->d_ctx_b, ctx->d_ctx_key1, ctx->d_ctx_key2 ));
	}
	else if(miner_algo == cryptonight_haven)
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_extra_gpu_prepare<cryptonight_haven><<<grid, block >>>( wsize, ctx->d_input, ctx->inputlen, startNonce,
			ctx->d_ctx_state,ctx->d_ctx_state2, ctx->d_ctx_a, ctx->d_ctx_b, ctx->d_ctx_key1, ctx->d_ctx_key2 ));
	}
	else if(miner_algo == cryptonight_superfast)
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_extra_gpu_prepare<cryptonight_superfast><<<grid, block >>>( wsize, ctx->d_input, ctx->inputlen, startNonce,
			ctx->d_ctx_state,ctx->d_ctx_state2, ctx->d_ctx_a, ctx->d_ctx_b, ctx->d_ctx_key1, ctx->d_ctx_key2 ));
	}
	else if(miner_algo == cryptonight_bittube2)
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_extra_gpu_prepare<cryptonight_bittube2><<<grid, block >>>( wsize, ctx->d_input, ctx->inputlen, startNonce,
			ctx->d_ctx_state,ctx->d_ctx_state2, ctx->d_ctx_a, ctx->d_ctx_b, ctx->d_ctx_key1, ctx->d_ctx_key2 ));
	}
	else if(miner_algo == cryptonight_monero_v8)
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_extra_gpu_prepare<cryptonight_monero_v8><<<grid, block >>>( wsize, ctx->d_input, ctx->inputlen, startNonce,
			ctx->d_ctx_state,ctx->d_ctx_state2, ctx->d_ctx_a, ctx->d_ctx_b, ctx->d_ctx_key1, ctx->d_ctx_key2 ));
	}
	else if(miner_algo == cryptonight_gpu)
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_extra_gpu_prepare<cryptonight_gpu><<<grid, block >>>( wsize, ctx->d_input, ctx->inputlen, startNonce,
			ctx->d_ctx_state,ctx->d_ctx_state2, ctx->d_ctx_a, ctx->d_ctx_b, ctx->d_ctx_key1, ctx->d_ctx_key2 ));
	}
	else if(miner_algo == cryptonight_r)
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_extra_gpu_prepare<cryptonight_r><<<grid, block >>>( wsize, ctx->d_input, ctx->inputlen, startNonce,
			ctx->d_ctx_state,ctx->d_ctx_state2, ctx->d_ctx_a, ctx->d_ctx_b, ctx->d_ctx_key1, ctx->d_ctx_key2 ));
	}
	else if(miner_algo == cryptonight_r_wow)
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_extra_gpu_prepare<cryptonight_r_wow><<<grid, block >>>( wsize, ctx->d_input, ctx->inputlen, startNonce,
			ctx->d_ctx_state,ctx->d_ctx_state2, ctx->d_ctx_a, ctx->d_ctx_b, ctx->d_ctx_key1, ctx->d_ctx_key2 ));
	}
	else if(miner_algo == cryptonight_v8_reversewaltz)
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_extra_gpu_prepare<cryptonight_v8_reversewaltz><<<grid, block >>>( wsize, ctx->d_input, ctx->inputlen, startNonce,
			ctx->d_ctx_state,ctx->d_ctx_state2, ctx->d_ctx_a, ctx->d_ctx_b, ctx->d_ctx_key1, ctx->d_ctx_key2 ));
	}
	else
	{
		/* pass two times d_ctx_state because the second state is used later in phase1,
		 * the first is used than in phase3
		 */
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_extra_gpu_prepare<invalid_algo><<<grid, block >>>( wsize, ctx->d_input, ctx->inputlen, startNonce,
			ctx->d_ctx_state, ctx->d_ctx_state, ctx->d_ctx_a, ctx->d_ctx_b, ctx->d_ctx_key1, ctx->d_ctx_key2 ));
	}
}

extern "C" void cryptonight_extra_cpu_final(nvid_ctx* ctx, uint32_t startNonce, uint64_t target, uint32_t* rescount, uint32_t *resnonce, const xmrstak_algo& miner_algo)
{
	int threadsperblock = 128;
	uint32_t wsize = ctx->device_blocks * ctx->device_threads;

	dim3 grid( ( wsize + threadsperblock - 1 ) / threadsperblock );
	dim3 block( threadsperblock );

	CUDA_CHECK(ctx->device_id, hipMemset( ctx->d_result_nonce, 0xFF, 10 * sizeof (uint32_t ) ));
	CUDA_CHECK(ctx->device_id, hipMemset( ctx->d_result_count, 0, sizeof (uint32_t ) ));

	if(miner_algo == cryptonight_heavy)
	{
		CUDA_CHECK_MSG_KERNEL(
			ctx->device_id,
			"\n**suggestion: Try to increase the value of the attribute 'bfactor' in the NVIDIA config file.**",
			cryptonight_extra_gpu_final<cryptonight_heavy><<<grid, block >>>( wsize, target, ctx->d_result_count, ctx->d_result_nonce, ctx->d_ctx_state,ctx->d_ctx_key2 )
		);
	}
	else if(miner_algo == cryptonight_haven)
	{
		CUDA_CHECK_MSG_KERNEL(
			ctx->device_id,
			"\n**suggestion: Try to increase the value of the attribute 'bfactor' in the NVIDIA config file.**",
			cryptonight_extra_gpu_final<cryptonight_haven><<<grid, block >>>( wsize, target, ctx->d_result_count, ctx->d_result_nonce, ctx->d_ctx_state,ctx->d_ctx_key2 )
		);
	}
	else if(miner_algo == cryptonight_superfast)
	{
		CUDA_CHECK_MSG_KERNEL(
			ctx->device_id,
			"\n**suggestion: Try to increase the value of the attribute 'bfactor' in the NVIDIA config file.**",
			cryptonight_extra_gpu_final<cryptonight_superfast><<<grid, block >>>( wsize, target, ctx->d_result_count, ctx->d_result_nonce, ctx->d_ctx_state,ctx->d_ctx_key2 )
		);
	}
	else if(miner_algo == cryptonight_bittube2)
	{
		CUDA_CHECK_MSG_KERNEL(
			ctx->device_id,
			"\n**suggestion: Try to increase the value of the attribute 'bfactor' in the NVIDIA config file.**",
			cryptonight_extra_gpu_final<cryptonight_bittube2><<<grid, block >>>( wsize, target, ctx->d_result_count, ctx->d_result_nonce, ctx->d_ctx_state,ctx->d_ctx_key2 )
		);
	}
	else if(miner_algo == cryptonight_gpu)
	{
		// fallback for all other algorithms
		CUDA_CHECK_MSG_KERNEL(
			ctx->device_id,
			"\n**suggestion: Try to increase the value of the attribute 'bfactor' in the NVIDIA config file.**",
			cryptonight_extra_gpu_final<cryptonight_gpu><<<grid, block >>>( wsize, target, ctx->d_result_count, ctx->d_result_nonce, ctx->d_ctx_state,ctx->d_ctx_key2 )
		);
	}
	else
	{
		// fallback for all other algorithms
		CUDA_CHECK_MSG_KERNEL(
			ctx->device_id,
			"\n**suggestion: Try to increase the value of the attribute 'bfactor' in the NVIDIA config file.**",
			cryptonight_extra_gpu_final<invalid_algo><<<grid, block >>>( wsize, target, ctx->d_result_count, ctx->d_result_nonce, ctx->d_ctx_state,ctx->d_ctx_key2 )
		);
	}

	CUDA_CHECK(ctx->device_id, hipMemcpy( rescount, ctx->d_result_count, sizeof (uint32_t ), hipMemcpyDeviceToHost ));
	CUDA_CHECK_MSG(
		ctx->device_id,
		"\n**suggestion: Try to increase the attribute 'bfactor' in the NVIDIA config file.**",
		hipMemcpy( resnonce, ctx->d_result_nonce, 10 * sizeof (uint32_t ), hipMemcpyDeviceToHost )
	);

	/* There is only a 32bit limit for the counter on the device side
	 * therefore this value can be greater than 10, in that case limit rescount
	 * to 10 entries.
	 */
	if(*rescount > 10)
		*rescount = 10;
	for(int i=0; i < *rescount; i++)
		resnonce[i] += startNonce;
}

extern "C" int cuda_get_devicecount( int* deviceCount)
{
	hipError_t err;
	*deviceCount = 0;
	err = hipGetDeviceCount(deviceCount);
	if(err != hipSuccess)
	{
		if(err == hipErrorNoDevice)
			printf("ERROR: NVIDIA no CUDA device found!\n");
		else if(err == hipErrorInsufficientDriver)
			printf("WARNING: NVIDIA Insufficient driver!\n");
		else
			printf("WARNING: NVIDIA Unable to query number of CUDA devices!\n");
		return 0;
	}

	return 1;
}

/** get device information
 *
 * @return 0 = all OK,
 *         1 = something went wrong,
 *         2 = gpu cannot be selected,
 *         3 = context cannot be created
 *         4 = not enough memory
 *         5 = architecture not supported (not compiled for the gpu architecture)
 */
extern "C" int cuda_get_deviceinfo(nvid_ctx* ctx)
{
	hipError_t err;
	int version;

	err = hipDriverGetVersion(&version);
	if(err != hipSuccess)
	{
		printf("Unable to query CUDA driver version! Is an nVidia driver installed?\n");
		return 1;
	}

	if(version < CUDART_VERSION)
	{
		printf("WARNING: Driver supports CUDA %d.%d but this was compiled for CUDA %d.%d API! Update your nVidia driver or compile with older CUDA!\n",
			version / 1000, (version % 1000 / 10),
			CUDART_VERSION / 1000, (CUDART_VERSION % 1000) / 10);
		return 1;
	}

	int GPU_N;
	if(cuda_get_devicecount(&GPU_N) == 0)
	{
		printf("WARNING: CUDA claims zero devices?\n");
		return 1;
	}

	if(ctx->device_id >= GPU_N)
	{
		printf("WARNING: Invalid device ID '%i'!\n", ctx->device_id);
		return 1;
	}

	hipDeviceProp_t props;
	err = hipGetDeviceProperties(&props, ctx->device_id);
	if(err != hipSuccess)
	{
		printf("\nGPU %d: %s\n%s line %d\n", ctx->device_id, hipGetErrorString(err), __FILE__, __LINE__);
		return 1;
	}

	ctx->device_name = strdup(props.name);
	ctx->device_mpcount = props.multiProcessorCount;
	ctx->device_arch[0] = props.major;
	ctx->device_arch[1] = props.minor;

	const int gpuArch = ctx->device_arch[0] * 10 + ctx->device_arch[1];

	ctx->name = std::string(props.name);

	printf("CUDA [%d.%d/%d.%d] GPU#%d, device architecture %d: \"%s\"... ",
		version / 1000, (version % 1000 / 10),
		CUDART_VERSION / 1000, (CUDART_VERSION % 1000) / 10,
		ctx->device_id, gpuArch, ctx->device_name);

	std::vector<int> arch;
#define XMRSTAK_PP_TOSTRING1(str) #str
#define XMRSTAK_PP_TOSTRING(str) XMRSTAK_PP_TOSTRING1(str)
	char const * archStringList = XMRSTAK_PP_TOSTRING(XMRSTAK_CUDA_ARCH_LIST);
#undef XMRSTAK_PP_TOSTRING
#undef XMRSTAK_PP_TOSTRING1
	std::stringstream ss(archStringList);

	//transform string list separated with `+` into a vector of integers
	int tmpArch;
	while ( ss >> tmpArch )
		arch.push_back( tmpArch );

	#define MSG_CUDA_NO_ARCH "WARNING: skip device - binary does not contain required device architecture\n"
	if(gpuArch >= 20 && gpuArch < 30)
	{
		// compiled binary must support sm_20 for fermi
		std::vector<int>::iterator it = std::find(arch.begin(), arch.end(), 20);
		if(it == arch.end())
		{
			printf(MSG_CUDA_NO_ARCH);
			return 5;
		}
	}
	if(gpuArch >= 30)
	{
		// search the minimum architecture greater than sm_20
		int minSupportedArch = 0;
		/* - for newer architecture than fermi we need at least sm_30
		 * or a architecture >= gpuArch
		 * - it is not possible to use a gpu with a architecture >= 30
		 *   with a sm_20 only compiled binary
		 */
		for(int i = 0; i < arch.size(); ++i)
			if(arch[i] >= 30  && (minSupportedArch == 0 || arch[i] < minSupportedArch))
				minSupportedArch = arch[i];
		if(minSupportedArch < 30 || gpuArch < minSupportedArch)
		{
			printf(MSG_CUDA_NO_ARCH);
			return 5;
		}
	}

	auto neededAlgorithms = ::jconf::inst()->GetCurrentCoinSelection().GetAllAlgorithms();
	bool useCryptonight_gpu = std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_gpu) != neededAlgorithms.end();


	// set all device option those marked as auto (-1) to a valid value
	if(ctx->device_blocks == -1)
	{
		/* good values based of my experience
		 *   - 3 * SMX count for >=sm_30
		 *   - 2 * SMX count for  <sm_30
		 */
		ctx->device_blocks = props.multiProcessorCount * (props.major < 3 ? 2 : 3);

		// use 6 blocks per SM for sm_2X else 8 blocks
		if(useCryptonight_gpu)
			ctx->device_blocks = props.multiProcessorCount * (props.major < 3 ? 6 : 8);

		// increase bfactor for low end devices to avoid that the miner is killed by the OS
		if(props.multiProcessorCount <= 6)
			ctx->device_bfactor += 2;
	}
	if(ctx->device_threads == -1)
	{
		/* sm_20 devices can only run 512 threads per cuda block
		 * `cryptonight_core_gpu_phase1` and `cryptonight_core_gpu_phase3` starts
		 * `8 * ctx->device_threads` threads per block
		 */
		const uint32_t maxThreadsPerBlock = props.major < 3 ? 512 : 1024;

		// for the most algorithms we are using 8 threads per hash
		uint32_t threadsPerHash = 8;

		// phase2_gpu uses 16 threads per hash
		if(useCryptonight_gpu)
			threadsPerHash = 16;

		ctx->device_threads = maxThreadsPerBlock / threadsPerHash;
		constexpr size_t byteToMiB = 1024u * 1024u;

		// no limit by default 1TiB
		size_t maxMemUsage = byteToMiB * byteToMiB;
		if(props.major == 6)
		{
			if(props.multiProcessorCount < 15)
			{
				// limit memory usage for GPUs for pascal < GTX1070
				maxMemUsage = size_t(2048u) * byteToMiB;
			}
			else if(props.multiProcessorCount <= 20)
			{
				// limit memory usage for GPUs for pascal GTX1070, GTX1080
				maxMemUsage = size_t(4096u) * byteToMiB;
			}
		}
		if(props.major < 6)
		{
			// limit memory usage for GPUs before pascal
			maxMemUsage = size_t(2048u) * byteToMiB;
		}
		if(props.major == 2)
		{
			// limit memory usage for sm 20 GPUs
			maxMemUsage = size_t(1024u) * byteToMiB;
		}

		if(props.multiProcessorCount <= 6)
		{
			// limit memory usage for low end devices to reduce the number of threads
			maxMemUsage = size_t(1024u) * byteToMiB;
		}

		int* tmp;
		hipError_t err;
		#define MSG_CUDA_FUNC_FAIL "WARNING: skip device - %s failed\n"
		// a device must be selected to get the right memory usage later on
		err = hipSetDevice(ctx->device_id);
		if(err != hipSuccess)
		{
			printf(MSG_CUDA_FUNC_FAIL, "hipSetDevice");
			return 2;
		}
		// trigger that a context on the gpu will be allocated
		err = hipMalloc(&tmp, 256);
		if(err != hipSuccess)
		{
			printf(MSG_CUDA_FUNC_FAIL, "hipMalloc");
			return 3;
		}


		size_t freeMemory = 0;
		size_t totalMemory = 0;
		CUDA_CHECK(ctx->device_id, hipMemGetInfo(&freeMemory, &totalMemory));

		CUDA_CHECK(ctx->device_id, hipFree(tmp));
		// delete created context on the gpu
		CUDA_CHECK(ctx->device_id, hipDeviceReset());

		ctx->total_device_memory = totalMemory;
		ctx->free_device_memory = freeMemory;

		size_t hashMemSize = 0;
		for(const auto algo : neededAlgorithms)
		{
			hashMemSize = std::max(hashMemSize, algo.Mem());
		}

#ifdef WIN32
		/* We use in windows bfactor (split slow kernel into smaller parts) to avoid
		 * that windows is killing long running kernel.
		 * In the case there is already memory used on the gpu than we
		 * assume that other application are running between the split kernel,
		 * this can result into TLB memory flushes and can strongly reduce the performance
		 * and the result can be that windows is killing the miner.
		 * Be reducing maxMemUsage we try to avoid this effect.
		 */
		size_t usedMem = totalMemory - freeMemory;
		if(usedMem >= maxMemUsage)
		{
			printf("WARNING: skip device - already %s MiB memory in use\n", std::to_string(usedMem/byteToMiB).c_str());
			return 4;
		}
		else
			maxMemUsage -= usedMem;

#endif
		// keep 128MiB memory free (value is randomly chosen)
		// 200byte are meta data memory (result nonce, ...)
		size_t availableMem = freeMemory - (128u * byteToMiB) - 200u;
		size_t limitedMemory = std::min(availableMem, maxMemUsage);
		// up to 16kibyte extra memory is used per thread for some kernel (lmem/local memory)
		// 680bytes are extra meta data memory per hash
		size_t perThread = hashMemSize + 16192u + 680u;
		if(
			std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_heavy) != neededAlgorithms.end() ||
			std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_haven) != neededAlgorithms.end() ||
			std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_bittube2) != neededAlgorithms.end() ||
			std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_superfast) != neededAlgorithms.end()
		)
			perThread += 50 * 4; // state double buffer

		size_t max_intensity = limitedMemory / perThread;
		ctx->device_threads = max_intensity / ctx->device_blocks;
		// use only odd number of threads
		ctx->device_threads = ctx->device_threads & 0xFFFFFFFE;

		if(ctx->device_threads > maxThreadsPerBlock / threadsPerHash)
		{
			ctx->device_threads = maxThreadsPerBlock / threadsPerHash;
		}

		// check if cryptonight_monero_v8 is selected for the user pool
		bool useCryptonight_v8 = (std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_monero_v8) != neededAlgorithms.end());
		bool useCryptonight_r = (std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_r) != neededAlgorithms.end());
		bool useCryptonight_r_wow = (std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_r_wow) != neededAlgorithms.end());
		bool useCryptonight_reversewaltz = (std::find(neededAlgorithms.begin(), neededAlgorithms.end(), cryptonight_v8_reversewaltz) != neededAlgorithms.end());

		// overwrite default config if cryptonight_monero_v8 is mined and GPU has at least compute capability 5.0
		if((useCryptonight_v8 || useCryptonight_r || useCryptonight_r_wow || useCryptonight_reversewaltz) && gpuArch >= 50)
		{
			// 4 based on my test maybe it must be adjusted later
			size_t threads = 4;
			// 8 is chosen by checking the occupancy calculator
			size_t blockOptimal = 8 * ctx->device_mpcount;

			if(blockOptimal * threads * hashMemSize < limitedMemory)
			{
				ctx->device_threads = threads;
				ctx->device_blocks = blockOptimal;
			}
		}
		else if(useCryptonight_gpu)
		{
			// 8 based on my profiling sessions maybe it must be adjusted later
			size_t threads = 8;
			// 8 is chosen by checking the occupancy calculator
			size_t blockOptimal = 8 * ctx->device_mpcount;

			// the following values are calculated with CUDA10 and the occupancy calculator
			if(gpuArch == 35 || gpuArch/10 == 5 || gpuArch/10 == 6)
				blockOptimal = 7 *  ctx->device_mpcount;
			if(gpuArch == 37)
				blockOptimal = 14 *  ctx->device_mpcount;
			if(gpuArch >= 70)
				blockOptimal = 6 *  ctx->device_mpcount;

			if(blockOptimal * threads * hashMemSize < limitedMemory)
			{
				ctx->device_threads = threads;
				ctx->device_blocks = blockOptimal;
			}

		}
	}
	printf("device init succeeded\n");

	return 0;
}
