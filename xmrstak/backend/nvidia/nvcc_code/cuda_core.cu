#include "hip/hip_runtime.h"
#include "xmrstak/backend/cryptonight.hpp"

#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "xmrstak/jconf.hpp"

#ifdef _WIN32
#include <windows.h>
extern "C" void compat_usleep(uint64_t waitTime)
{
    if (waitTime > 0)
    {
        if (waitTime > 100)
        {
            // use a waitable timer for larger intervals > 0.1ms

            HANDLE timer;
            LARGE_INTEGER ft;

            ft.QuadPart = -10ll * int64_t(waitTime); // Convert to 100 nanosecond interval, negative value indicates relative time

            timer = CreateWaitableTimer(NULL, TRUE, NULL);
            SetWaitableTimer(timer, &ft, 0, NULL, NULL, 0);
            WaitForSingleObject(timer, INFINITE);
            CloseHandle(timer);
        }
        else
        {
            // use a polling loop for short intervals <= 100ms

            LARGE_INTEGER perfCnt, start, now;
            __int64 elapsed;

            QueryPerformanceFrequency(&perfCnt);
            QueryPerformanceCounter(&start);
            do {
		SwitchToThread();
                QueryPerformanceCounter((LARGE_INTEGER*) &now);
                elapsed = (__int64)((now.QuadPart - start.QuadPart) / (float)perfCnt.QuadPart * 1000 * 1000);
            } while ( elapsed < waitTime );
        }
    }
}
#else
#include <unistd.h>
extern "C" void compat_usleep(uint64_t waitTime)
{
	usleep(waitTime);
}
#endif

#include "cryptonight.hpp"
#include "cuda_extra.hpp"
#include "cuda_aes.hpp"
#include "cuda_device.hpp"

/* sm_2X is limited to 2GB due to the small TLB
 * therefore we never use 64bit indices
 */
#if defined(XMR_STAK_LARGEGRID) && (__CUDA_ARCH__ >= 300)
typedef uint64_t IndexType;
#else
typedef int IndexType;
#endif

__device__ __forceinline__ uint64_t cuda_mul128( uint64_t multiplier, uint64_t multiplicand, uint64_t* product_hi )
{
	*product_hi = __umul64hi( multiplier, multiplicand );
	return (multiplier * multiplicand );
}

template< typename T >
__device__ __forceinline__ T loadGlobal64( T * const addr )
{
#if (__CUDA_ARCH__ < 700)
	T x;
	asm volatile( "ld.global.cg.u64 %0, [%1];" : "=l"( x ) : "l"( addr ) );
	return x;
#else
	return *addr;
#endif
}

template< typename T >
__device__ __forceinline__ T loadGlobal32( T * const addr )
{
#if (__CUDA_ARCH__ < 700)
	T x;
	asm volatile( "ld.global.cg.u32 %0, [%1];" : "=r"( x ) : "l"( addr ) );
	return x;
#else
	return *addr;
#endif
}


template< typename T >
__device__ __forceinline__ void storeGlobal32( T* addr, T const & val )
{
#if (__CUDA_ARCH__ < 700)
	asm volatile( "st.global.cg.u32 [%0], %1;" : : "l"( addr ), "r"( val ) );
#else
	*addr = val;
#endif
}

template< typename T >
__device__ __forceinline__ void storeGlobal64( T* addr, T const & val )
{
#if (__CUDA_ARCH__ < 700)
	asm volatile( "st.global.cg.u64 [%0], %1;" : : "l"( addr ), "l"( val ) );
#else
	*addr = val;
#endif
}

template<size_t ITERATIONS, uint32_t MEMORY>
__global__ void cryptonight_core_gpu_phase1( int threads, int bfactor, int partidx, uint32_t * __restrict__ long_state, uint32_t * __restrict__ ctx_state2, uint32_t * __restrict__ ctx_key1 )
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init( sharedMemory );
	__syncthreads( );

	const int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 3;
	const int sub = ( threadIdx.x & 7 ) << 2;

	const int batchsize = MEMORY >> bfactor;
	const int start = partidx * batchsize;
	const int end = start + batchsize;

	if ( thread >= threads )
		return;

	uint32_t key[40], text[4];

	MEMCPY8( key, ctx_key1 + thread * 40, 20 );

	if( partidx == 0 )
	{
		// first round
		MEMCPY8( text, ctx_state2 + thread * 50 + sub + 16, 2 );
	}
	else
	{
		// load previous text data
		MEMCPY8( text, &long_state[( (uint64_t) thread * MEMORY ) + sub + start - 32], 2 );
	}
	__syncthreads( );
	for ( int i = start; i < end; i += 32 )
	{
		cn_aes_pseudo_round_mut( sharedMemory, text, key );
		MEMCPY8(&long_state[((uint64_t) thread * MEMORY) + (sub + i)], text, 2);
	}
}

/** avoid warning `unused parameter` */
template< typename T >
__forceinline__ __device__ void unusedVar( const T& )
{
}

/** shuffle data for
 *
 * - this method can be used with all compute architectures
 * - for <sm_30 shared memory is needed
 *
 * group_n - must be a power of 2!
 * 
 * @param ptr pointer to shared memory, size must be `threadIdx.x * sizeof(uint32_t)`
 *            value can be NULL for compute architecture >=sm_30
 * @param sub thread number within the group, range [0:group_n]
 * @param value value to share with other threads within the group
 * @param src thread number within the group from where the data is read, range [0:group_n]
 */
template<size_t group_n>
__forceinline__ __device__ uint32_t shuffle(volatile uint32_t* ptr,const uint32_t sub,const int val,const uint32_t src)
{
#if( __CUDA_ARCH__ < 300 )
    ptr[sub] = val;
    return ptr[src & (group_n-1)];
#else
    unusedVar( ptr );
    unusedVar( sub );
#   if(__CUDACC_VER_MAJOR__ >= 9)
    return __shfl_sync(0xFFFFFFFF, val, src, group_n );
#	else
	return __shfl( val, src, group_n );
#	endif
#endif
}

template<size_t ITERATIONS, uint32_t MEMORY, uint32_t MASK, xmrstak_algo ALGO>
#ifdef XMR_STAK_THREADS
__launch_bounds__( XMR_STAK_THREADS * 4 )
#endif
__global__ void cryptonight_core_gpu_phase2( int threads, int bfactor, int partidx, uint32_t * d_long_state, uint32_t * d_ctx_a, uint32_t * d_ctx_b, uint32_t * d_ctx_state,
		uint32_t startNonce, uint32_t * __restrict__ d_input )
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init( sharedMemory );

	__syncthreads( );

	const int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 2;
	const uint32_t nonce = startNonce + thread;
	const int sub = threadIdx.x & 3;
	const int sub2 = sub & 2;

#if( __CUDA_ARCH__ < 300 )
        extern __shared__ uint32_t shuffleMem[];
        volatile uint32_t* sPtr = (volatile uint32_t*)(shuffleMem + (threadIdx.x& 0xFFFFFFFC));
#else
        volatile uint32_t* sPtr = NULL;
#endif
	if ( thread >= threads )
		return;

	int i, k;
	uint32_t j;
	const int batchsize = (ITERATIONS * 2) >> ( 2 + bfactor );
	const int start = partidx * batchsize;
	const int end = start + batchsize;
	uint32_t * long_state = &d_long_state[(IndexType) thread * MEMORY];
	uint32_t a, d[2], idx0;
	uint32_t t1[2], t2[2], res;

	uint32_t tweak1_2[2];
	if (ALGO == cryptonight_monero || ALGO == cryptonight_aeon)
	{
		uint32_t * state = d_ctx_state + thread * 50;
		tweak1_2[0] = (d_input[8] >> 24) | (d_input[9] << 8);
		tweak1_2[0] ^= state[48];
		tweak1_2[1] = nonce;
		tweak1_2[1] ^= state[49];
	}

	a = (d_ctx_a + thread * 4)[sub];
	idx0 = shuffle<4>(sPtr,sub, a, 0);
	if(ALGO == cryptonight_heavy)
	{
		if(partidx != 0)
		{
			// state is stored after all ctx_b states
			idx0 = *(d_ctx_b + threads * 4 + thread);
		}
	}
	d[1] = (d_ctx_b + thread * 4)[sub];

	#pragma unroll 2
	for ( i = start; i < end; ++i )
	{
		#pragma unroll 2
		for ( int x = 0; x < 2; ++x )
		{
			j = ( ( idx0 & MASK ) >> 2 ) + sub;

			const uint32_t x_0 = loadGlobal32<uint32_t>( long_state + j );
			const uint32_t x_1 = shuffle<4>(sPtr,sub, x_0, sub + 1);
			const uint32_t x_2 = shuffle<4>(sPtr,sub, x_0, sub + 2);
			const uint32_t x_3 = shuffle<4>(sPtr,sub, x_0, sub + 3);
			d[x] = a ^
				t_fn0( x_0 & 0xff ) ^
				t_fn1( (x_1 >> 8) & 0xff ) ^
				t_fn2( (x_2 >> 16) & 0xff ) ^
				t_fn3( ( x_3 >> 24 ) );


			//XOR_BLOCKS_DST(c, b, &long_state[j]);
			t1[0] = shuffle<4>(sPtr,sub, d[x], 0);

			const uint32_t z = d[0] ^ d[1];
			if(ALGO == cryptonight_monero || ALGO == cryptonight_aeon)
			{
				const uint32_t table = 0x75310U;
				const uint32_t index = ((z >> 26) & 12) | ((z >> 23) & 2);
				const uint32_t fork_7 = z ^ ((table >> index) & 0x30U) << 24;
				storeGlobal32( long_state + j, sub == 2 ? fork_7 : z );
			}
			else
				storeGlobal32( long_state + j, z );
			
			//MUL_SUM_XOR_DST(c, a, &long_state[((uint32_t *)c)[0] & MASK]);
			j = ( ( *t1 & MASK ) >> 2 ) + sub;

			uint32_t yy[2];
			*( (uint64_t*) yy ) = loadGlobal64<uint64_t>( ( (uint64_t *) long_state )+( j >> 1 ) );
			uint32_t zz[2];
			zz[0] = shuffle<4>(sPtr,sub, yy[0], 0);
			zz[1] = shuffle<4>(sPtr,sub, yy[1], 0);

			t1[1] = shuffle<4>(sPtr,sub, d[x], 1);
			#pragma unroll
			for ( k = 0; k < 2; k++ )
				t2[k] = shuffle<4>(sPtr,sub, a, k + sub2);

            *( (uint64_t *) t2 ) += sub2 ? ( *( (uint64_t *) t1 ) * *( (uint64_t*) zz ) ) : __umul64hi( *( (uint64_t *) t1 ), *( (uint64_t*) zz ) );

			res = *( (uint64_t *) t2 )  >> ( sub & 1 ? 32 : 0 );

			
			if(ALGO == cryptonight_monero || ALGO == cryptonight_aeon)
			{
				const uint32_t tweaked_res = tweak1_2[sub & 1] ^ res;
				const uint32_t long_state_update = sub2 ? tweaked_res : res;
				storeGlobal32( long_state + j, long_state_update );
			}
			else
				storeGlobal32( long_state + j, res );
			
			a = ( sub & 1 ? yy[1] : yy[0] ) ^ res;
			idx0 = shuffle<4>(sPtr,sub, a, 0);
			if(ALGO == cryptonight_heavy)
			{
				int64_t n = loadGlobal64<uint64_t>( ( (uint64_t *) long_state ) + (( idx0 & MASK ) >> 3));
				int32_t d = loadGlobal32<uint32_t>( (uint32_t*)(( (uint64_t *) long_state ) + (( idx0 & MASK) >> 3) + 1u ));
				int64_t q = n / (d | 0x5);

				if(sub&1)
					storeGlobal64<uint64_t>( ( (uint64_t *) long_state ) + (( idx0 & MASK ) >> 3), n ^ q );

				idx0 = d ^ q;
			}
		}
	}

	if ( bfactor > 0 )
	{
		(d_ctx_a + thread * 4)[sub] = a;
		(d_ctx_b + thread * 4)[sub] = d[1];
		if(ALGO == cryptonight_heavy)
			if(sub&1)
				*(d_ctx_b + threads * 4 + thread) = idx0;
	}
}

template<size_t ITERATIONS, uint32_t MEMORY, xmrstak_algo ALGO>
__global__ void cryptonight_core_gpu_phase3( int threads, int bfactor, int partidx, const uint32_t * __restrict__ long_state, uint32_t * __restrict__ d_ctx_state, uint32_t * __restrict__ d_ctx_key2 )
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init( sharedMemory );
	__syncthreads( );

	int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 3;
	int subv = ( threadIdx.x & 7 );
	int sub = subv << 2;

	const int batchsize = MEMORY >> bfactor;
	const int start = (partidx % (1 << bfactor)) * batchsize;
	const int end = start + batchsize;

	if ( thread >= threads )
		return;

	uint32_t key[40], text[4];
	MEMCPY8( key, d_ctx_key2 + thread * 40, 20 );
	MEMCPY8( text, d_ctx_state + thread * 50 + sub + 16, 2 );

	__syncthreads( );
	
#if( __CUDA_ARCH__ < 300 )
	extern __shared__ uint32_t shuffleMem[];
	volatile uint32_t* sPtr = (volatile uint32_t*)(shuffleMem + (threadIdx.x& 0xFFFFFFF8));
#else
	volatile uint32_t* sPtr = NULL;
#endif

	for ( int i = start; i < end; i += 32 )
	{
		#pragma unroll
		for ( int j = 0; j < 4; ++j )
			text[j] ^= long_state[((IndexType) thread * MEMORY) + ( sub + i + j)];

		cn_aes_pseudo_round_mut( sharedMemory, text, key );
		
		if(ALGO == cryptonight_heavy)
		{
			#pragma unroll
			for ( int j = 0; j < 4; ++j )
				text[j] ^= shuffle<8>(sPtr, subv, text[j], (subv+1)&7);
		}
	}

	MEMCPY8( d_ctx_state + thread * 50 + sub + 16, text, 2 );
}

template<size_t ITERATIONS, uint32_t MASK, uint32_t MEMORY, xmrstak_algo ALGO>
void cryptonight_core_gpu_hash(nvid_ctx* ctx, uint32_t nonce)
{
	dim3 grid( ctx->device_blocks );
	dim3 block( ctx->device_threads );
	dim3 block4( ctx->device_threads << 2 );
	dim3 block8( ctx->device_threads << 3 );

	int partcount = 1 << ctx->device_bfactor;

	/* bfactor for phase 1 and 3
	 *
	 * phase 1 and 3 consume less time than phase 2, therefore we begin with the
	 * kernel splitting if the user defined a `bfactor >= 5`
	 */
	int bfactorOneThree = ctx->device_bfactor - 4;
	if( bfactorOneThree < 0 )
		bfactorOneThree = 0;

	int partcountOneThree = 1 << bfactorOneThree;

	for ( int i = 0; i < partcountOneThree; i++ )
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase1<ITERATIONS,MEMORY><<< grid, block8 >>>( ctx->device_blocks*ctx->device_threads,
			bfactorOneThree, i,
			ctx->d_long_state, 
			(ALGO == cryptonight_heavy ? ctx->d_ctx_state2 : ctx->d_ctx_state),
			ctx->d_ctx_key1 ));

		if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );
	}
	if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );

	for ( int i = 0; i < partcount; i++ )
	{
        CUDA_CHECK_MSG_KERNEL(
			ctx->device_id,
			"\n**suggestion: Try to increase the value of the attribute 'bfactor' or \nreduce 'threads' in the NVIDIA config file.**",
			cryptonight_core_gpu_phase2<ITERATIONS,MEMORY,MASK,ALGO><<<
				grid,
				block4,
				block4.x * sizeof(uint32_t) * static_cast< int >( ctx->device_arch[0] < 3 )
			>>>(
				ctx->device_blocks*ctx->device_threads,
				ctx->device_bfactor,
				i,
				ctx->d_long_state,
				ctx->d_ctx_a,
				ctx->d_ctx_b,
				ctx->d_ctx_state,
				nonce,
				ctx->d_input
			)
	    );

		if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );
	}

	int roundsPhase3 = partcountOneThree;

	if(ALGO == cryptonight_heavy)
	{
		// cryptonight_heavy used two full rounds over the scratchpad memory
		roundsPhase3 *= 2;
	}

	for ( int i = 0; i < roundsPhase3; i++ )
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase3<ITERATIONS,MEMORY, ALGO><<< 
			grid,
			block8,
			block8.x * sizeof(uint32_t) * static_cast< int >( ctx->device_arch[0] < 3 )
		>>>( ctx->device_blocks*ctx->device_threads,
			bfactorOneThree, i,
			ctx->d_long_state,
			ctx->d_ctx_state, ctx->d_ctx_key2 ));
	}
}

void cryptonight_core_cpu_hash(nvid_ctx* ctx, xmrstak_algo miner_algo, uint32_t startNonce)
{

	if(miner_algo == cryptonight_monero)
	{
		cryptonight_core_gpu_hash<CRYPTONIGHT_ITER, CRYPTONIGHT_MASK, CRYPTONIGHT_MEMORY/4, cryptonight_monero>(ctx, startNonce);
	}
	else if(miner_algo == cryptonight_heavy)
	{
		cryptonight_core_gpu_hash<CRYPTONIGHT_HEAVY_ITER, CRYPTONIGHT_HEAVY_MASK, CRYPTONIGHT_HEAVY_MEMORY/4, cryptonight_heavy>(ctx, startNonce);
	}
	else if(miner_algo == cryptonight)
	{
		cryptonight_core_gpu_hash<CRYPTONIGHT_ITER, CRYPTONIGHT_MASK, CRYPTONIGHT_MEMORY/4, cryptonight>(ctx, startNonce);
	}
	else if(miner_algo == cryptonight_lite)
	{
		cryptonight_core_gpu_hash<CRYPTONIGHT_LITE_ITER, CRYPTONIGHT_LITE_MASK, CRYPTONIGHT_LITE_MEMORY/4, cryptonight_lite>(ctx, startNonce);
	}
	else if(miner_algo == cryptonight_aeon)
	{
		cryptonight_core_gpu_hash<CRYPTONIGHT_LITE_ITER, CRYPTONIGHT_LITE_MASK, CRYPTONIGHT_LITE_MEMORY/4, cryptonight_aeon>(ctx, startNonce);
	}
else if(miner_algo == cryptonight_alloy)
	{
		cryptonight_core_gpu_hash<CRYPTONIGHT_ALLOY_ITER, CRYPTONIGHT_ALLOY_MASK, CRYPTONIGHT_ALLOY_MEMORY/4, cryptonight_alloy>(ctx, startNonce);
	}
}
