#include "hip/hip_runtime.h"
#include "xmrstak/backend/cryptonight.hpp"

#include <stdio.h>
#include <stdint.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#ifdef _WIN32
#include <windows.h>
extern "C" void compat_usleep(uint64_t waitTime)
{
    if (waitTime > 0)
    {
        if (waitTime > 100)
        {
            // use a waitable timer for larger intervals > 0.1ms

            HANDLE timer;
            LARGE_INTEGER ft;

            ft.QuadPart = -10ll * int64_t(waitTime); // Convert to 100 nanosecond interval, negative value indicates relative time

            timer = CreateWaitableTimer(NULL, TRUE, NULL);
            SetWaitableTimer(timer, &ft, 0, NULL, NULL, 0);
            WaitForSingleObject(timer, INFINITE);
            CloseHandle(timer);
        }
        else
        {
            // use a polling loop for short intervals <= 100ms

            LARGE_INTEGER perfCnt, start, now;
            __int64 elapsed;

            QueryPerformanceFrequency(&perfCnt);
            QueryPerformanceCounter(&start);
            do {
		SwitchToThread();
                QueryPerformanceCounter((LARGE_INTEGER*) &now);
                elapsed = (__int64)((now.QuadPart - start.QuadPart) / (float)perfCnt.QuadPart * 1000 * 1000);
            } while ( elapsed < waitTime );
        }
    }
}
#else
#include <unistd.h>
extern "C" void compat_usleep(uint64_t waitTime)
{
	usleep(waitTime);
}
#endif

#include "cryptonight.hpp"
#include "cuda_extra.hpp"
#include "cuda_aes.hpp"
#include "cuda_device.hpp"

/* sm_2X is limited to 2GB due to the small TLB
 * therefore we never use 64bit indices
 */
#if defined(XMR_STAK_LARGEGRID) && (__CUDA_ARCH__ >= 300)
typedef uint64_t IndexType;
#else
typedef int IndexType;
#endif

__device__ __forceinline__ uint64_t cuda_mul128( uint64_t multiplier, uint64_t multiplicand, uint64_t* product_hi )
{
	*product_hi = __umul64hi( multiplier, multiplicand );
	return (multiplier * multiplicand );
}

template< typename T >
__device__ __forceinline__ T loadGlobal64( T * const addr )
{
	T x;
	asm volatile( "ld.global.cg.u64 %0, [%1];" : "=l"( x ) : "l"( addr ) );
	return x;
}

template< typename T >
__device__ __forceinline__ T loadGlobal32( T * const addr )
{
	T x;
	asm volatile( "ld.global.cg.u32 %0, [%1];" : "=r"( x ) : "l"( addr ) );
	return x;
}


template< typename T >
__device__ __forceinline__ void storeGlobal32( T* addr, T const & val )
{
	asm volatile( "st.global.cg.u32 [%0], %1;" : : "l"( addr ), "r"( val ) );
}

template<size_t ITERATIONS, uint32_t THREAD_SHIFT>
__global__ void cryptonight_core_gpu_phase1( int threads, int bfactor, int partidx, uint32_t * __restrict__ long_state, uint32_t * __restrict__ ctx_state, uint32_t * __restrict__ ctx_key1 )
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init( sharedMemory );
	__syncthreads( );

	const int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 3;
	const int sub = ( threadIdx.x & 7 ) << 2;

	const int batchsize = ITERATIONS >> bfactor;
	const int start = partidx * batchsize;
	const int end = start + batchsize;

	if ( thread >= threads )
		return;

	uint32_t key[40], text[4];

	MEMCPY8( key, ctx_key1 + thread * 40, 20 );

	if( partidx == 0 )
	{
		// first round
		MEMCPY8( text, ctx_state + thread * 50 + sub + 16, 2 );
	}
	else
	{
		// load previous text data
		MEMCPY8( text, &long_state[( (uint64_t) thread << THREAD_SHIFT ) + sub + start - 32], 2 );
	}
	__syncthreads( );
	for ( int i = start; i < end; i += 32 )
	{
		cn_aes_pseudo_round_mut( sharedMemory, text, key );
		MEMCPY8(&long_state[((uint64_t) thread << THREAD_SHIFT) + (sub + i)], text, 2);
	}
}

/** avoid warning `unused parameter` */
template< typename T >
__forceinline__ __device__ void unusedVar( const T& )
{
}

/** shuffle data for
 *
 * - this method can be used with all compute architectures
 * - for <sm_30 shared memory is needed
 *
 * @param ptr pointer to shared memory, size must be `threadIdx.x * sizeof(uint32_t)`
 *            value can be NULL for compute architecture >=sm_30
 * @param sub thread number within the group, range [0;4)
 * @param value value to share with other threads within the group
 * @param src thread number within the group from where the data is read, range [0;4)
 */
__forceinline__ __device__ uint32_t shuffle(volatile uint32_t* ptr,const uint32_t sub,const int val,const uint32_t src)
{
#if( __CUDA_ARCH__ < 300 )
    ptr[sub] = val;
    return ptr[src&3];
#else
    unusedVar( ptr );
    unusedVar( sub );
#   if(__CUDACC_VER_MAJOR__ >= 9)
    return __shfl_sync(0xFFFFFFFF, val, src, 4 );
#	else
	return __shfl( val, src, 4 );
#	endif
#endif
}

template<size_t ITERATIONS, uint32_t THREAD_SHIFT, uint32_t MASK>
#ifdef XMR_STAK_THREADS
__launch_bounds__( XMR_STAK_THREADS * 4 )
#endif
__global__ void cryptonight_core_gpu_phase2( int threads, int bfactor, int partidx, uint32_t * d_long_state, uint32_t * d_ctx_a, uint32_t * d_ctx_b )
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init( sharedMemory );

	__syncthreads( );

	const int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 2;
	const int sub = threadIdx.x & 3;
	const int sub2 = sub & 2;

#if( __CUDA_ARCH__ < 300 )
        extern __shared__ uint32_t shuffleMem[];
        volatile uint32_t* sPtr = (volatile uint32_t*)(shuffleMem + (threadIdx.x& 0xFFFFFFFC));
#else
        volatile uint32_t* sPtr = NULL;
#endif
	if ( thread >= threads )
		return;

	int i, k;
        uint32_t j;
	const int batchsize = (ITERATIONS * 2) >> ( 2 + bfactor );
	const int start = partidx * batchsize;
	const int end = start + batchsize;
	uint32_t * long_state = &d_long_state[(IndexType) thread << THREAD_SHIFT];
	uint32_t * ctx_a = d_ctx_a + thread * 4;
	uint32_t * ctx_b = d_ctx_b + thread * 4;
	uint32_t a, d[2];
	uint32_t t1[2], t2[2], res;

	a = ctx_a[sub];
	d[1] = ctx_b[sub];
	#pragma unroll 2
	for ( i = start; i < end; ++i )
	{
		#pragma unroll 2
		for ( int x = 0; x < 2; ++x )
		{
			j = ( ( shuffle(sPtr,sub, a, 0) & MASK ) >> 2 ) + sub;

			const uint32_t x_0 = loadGlobal32<uint32_t>( long_state + j );
			const uint32_t x_1 = shuffle(sPtr,sub, x_0, sub + 1);
			const uint32_t x_2 = shuffle(sPtr,sub, x_0, sub + 2);
			const uint32_t x_3 = shuffle(sPtr,sub, x_0, sub + 3);
			d[x] = a ^
				t_fn0( x_0 & 0xff ) ^
				t_fn1( (x_1 >> 8) & 0xff ) ^
				t_fn2( (x_2 >> 16) & 0xff ) ^
				t_fn3( ( x_3 >> 24 ) );


			//XOR_BLOCKS_DST(c, b, &long_state[j]);
			t1[0] = shuffle(sPtr,sub, d[x], 0);
			//long_state[j] = d[0] ^ d[1];
			storeGlobal32( long_state + j, d[0] ^ d[1] );

			//MUL_SUM_XOR_DST(c, a, &long_state[((uint32_t *)c)[0] & MASK]);
			j = ( ( *t1 & MASK ) >> 2 ) + sub;

			uint32_t yy[2];
			*( (uint64_t*) yy ) = loadGlobal64<uint64_t>( ( (uint64_t *) long_state )+( j >> 1 ) );
			uint32_t zz[2];
			zz[0] = shuffle(sPtr,sub, yy[0], 0);
			zz[1] = shuffle(sPtr,sub, yy[1], 0);

			t1[1] = shuffle(sPtr,sub, d[x], 1);
			#pragma unroll
			for ( k = 0; k < 2; k++ )
				t2[k] = shuffle(sPtr,sub, a, k + sub2);

            *( (uint64_t *) t2 ) += sub2 ? ( *( (uint64_t *) t1 ) * *( (uint64_t*) zz ) ) : __umul64hi( *( (uint64_t *) t1 ), *( (uint64_t*) zz ) );

			res = *( (uint64_t *) t2 )  >> ( sub & 1 ? 32 : 0 );

			storeGlobal32( long_state + j, res );
			a = ( sub & 1 ? yy[1] : yy[0] ) ^ res;
		}
	}

	if ( bfactor > 0 )
	{
		ctx_a[sub] = a;
		ctx_b[sub] = d[1];
	}
}

template<size_t ITERATIONS, uint32_t THREAD_SHIFT>
__global__ void cryptonight_core_gpu_phase3( int threads, int bfactor, int partidx, const uint32_t * __restrict__ long_state, uint32_t * __restrict__ d_ctx_state, uint32_t * __restrict__ d_ctx_key2 )
{
	__shared__ uint32_t sharedMemory[1024];

	cn_aes_gpu_init( sharedMemory );
	__syncthreads( );

	int thread = ( blockDim.x * blockIdx.x + threadIdx.x ) >> 3;
	int sub = ( threadIdx.x & 7 ) << 2;

	const int batchsize = ITERATIONS >> bfactor;
	const int start = partidx * batchsize;
	const int end = start + batchsize;

	if ( thread >= threads )
		return;

	uint32_t key[40], text[4];
	MEMCPY8( key, d_ctx_key2 + thread * 40, 20 );
	MEMCPY8( text, d_ctx_state + thread * 50 + sub + 16, 2 );

	__syncthreads( );
	for ( int i = start; i < end; i += 32 )
	{
#pragma unroll
		for ( int j = 0; j < 4; ++j )
			text[j] ^= long_state[((IndexType) thread << THREAD_SHIFT) + (sub + i + j)];

		cn_aes_pseudo_round_mut( sharedMemory, text, key );
	}

	MEMCPY8( d_ctx_state + thread * 50 + sub + 16, text, 2 );
}

template<size_t ITERATIONS, uint32_t MASK, uint32_t THREAD_SHIFT>
void cryptonight_core_gpu_hash(nvid_ctx* ctx)
{
	dim3 grid( ctx->device_blocks );
	dim3 block( ctx->device_threads );
	dim3 block4( ctx->device_threads << 2 );
	dim3 block8( ctx->device_threads << 3 );

	int partcount = 1 << ctx->device_bfactor;

	/* bfactor for phase 1 and 3
	 *
	 * phase 1 and 3 consume less time than phase 2, therefore we begin with the
	 * kernel splitting if the user defined a `bfactor >= 5`
	 */
	int bfactorOneThree = ctx->device_bfactor - 4;
	if( bfactorOneThree < 0 )
		bfactorOneThree = 0;

	int partcountOneThree = 1 << bfactorOneThree;

	for ( int i = 0; i < partcountOneThree; i++ )
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase1<ITERATIONS,THREAD_SHIFT><<< grid, block8 >>>( ctx->device_blocks*ctx->device_threads,
			bfactorOneThree, i,
			ctx->d_long_state, ctx->d_ctx_state, ctx->d_ctx_key1 ));

		if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );
	}
	if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );

	for ( int i = 0; i < partcount; i++ )
	{
        CUDA_CHECK_MSG_KERNEL(
			ctx->device_id,
			"\n**suggestion: Try to increase the value of the attribute 'bfactor' or \nreduce 'threads' in the NVIDIA config file.**",
			cryptonight_core_gpu_phase2<ITERATIONS,THREAD_SHIFT,MASK><<<
				grid,
				block4,
				block4.x * sizeof(uint32_t) * static_cast< int >( ctx->device_arch[0] < 3 )
			>>>(
				ctx->device_blocks*ctx->device_threads,
				ctx->device_bfactor,
				i,
				ctx->d_long_state,
				ctx->d_ctx_a,
				ctx->d_ctx_b
			)
	    );

		if ( partcount > 1 && ctx->device_bsleep > 0) compat_usleep( ctx->device_bsleep );
	}

	for ( int i = 0; i < partcountOneThree; i++ )
	{
		CUDA_CHECK_KERNEL(ctx->device_id, cryptonight_core_gpu_phase3<ITERATIONS,THREAD_SHIFT><<< grid, block8 >>>( ctx->device_blocks*ctx->device_threads,
			bfactorOneThree, i,
			ctx->d_long_state,
			ctx->d_ctx_state, ctx->d_ctx_key2 ));
	}
}

void cryptonight_core_cpu_hash(nvid_ctx* ctx, bool mineMonero)
{
#ifndef CONF_NO_MONERO
	if(mineMonero)
	{
		cryptonight_core_gpu_hash<MONERO_ITER, MONERO_MASK, 19u>(ctx);
	}
#endif
#ifndef CONF_NO_AEON
	if(!mineMonero)
	{
		cryptonight_core_gpu_hash<AEON_ITER, AEON_MASK, 18u>(ctx);
	}
#endif
}
