/*
Copyright (c) 2019 SChernykh

This file is part of RandomX CUDA.

RandomX CUDA is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

RandomX CUDA is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with RandomX CUDA.  If not, see<http://www.gnu.org/licenses/>.
*/

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <chrono>
#include <thread>
#include "xmrstak/backend/nvidia/nvcc_code/cryptonight.hpp"
#include "../nvcc_code/cuda_device.hpp"

namespace RandomX_Coinevo {
    #include "configuration.h"
    #define fillAes4Rx4 fillAes4Rx4_v104
    #include "../RandomX/common.hpp"
}
